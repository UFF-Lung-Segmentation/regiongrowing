#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define IMAGE_INPUT_DIR "dataset/Cx2_Ima1/csv"
#define IMAGE_OUTPUT_DIR "dataset/Cx2_Ima1/region"
#define WIDTH 512
#define MAX_NUMBER_CORTES 300

// Limiar area dos pulmões
#define HU_PULMAO_MIN -700
#define HU_PULMAO_MAX -600
      
// MIN/MAX das tomografias dos ratos
#define MIN_HU -1024
#define MAX_HU 100

// número de features extraídas de cada voxel
#define NUM_FEATURES 5
#define LIMIAR 0.9

// constantes para trabalhar com arqivos
#define MAX_LINE_SIZE 3072 // se pixel value 16bit: valores -32768 a +32767: 6 caracteres * 512 elementos por linha = 3.072 
#define MAX_TOKEN_SIZE 6
#define MAX_FILENAME 1024  


// *********************************************************************
// funcao que informa a quantidade de cortes no diretório
// *********************************************************************
int countSlices(){
  DIR *d;
  struct dirent *dir;
  d = opendir(IMAGE_INPUT_DIR);
  int num_slices = 0;
  if (d){
      while ((dir = readdir(d)) != NULL)
      {
        if (dir->d_type == DT_REG){
          num_slices++;
        }
      }
      closedir(d);
  } else {
    printf("countSlices: não conseguiu ler o diretório\n");
    return(-1);
  }
  return num_slices;
}

// *********************************************************************
// funcao para carregar os cortes do filesystem para a memória principal
// *********************************************************************
int loadCT(int *imagem){

  
  // verifica os arquivos no diretorio
  DIR *d;
  struct dirent *dir;
  d = opendir(IMAGE_INPUT_DIR);
  char files[MAX_NUMBER_CORTES][MAX_FILENAME];
  int num_files = 0;
  if (d){
      while ((dir = readdir(d)) != NULL)
      {
        if (dir->d_type == DT_REG){
          char filename[MAX_FILENAME] = IMAGE_INPUT_DIR "/";
          strcpy(files[num_files++], strcat(filename,dir->d_name));
        }
      }
      closedir(d);
  } else {
    printf("loadCT: não conseguiu ler o diretório\n");
    return(-1);
  }

  // ordena a lista de arquivos
  for (int i = 0; i < num_files; i++){
    for (int j = 0; j < num_files; j++){      
      if (strcmp(files[i], files[j]) < 0){
        char temp[MAX_FILENAME] = {};
        strcpy(temp,files[i]);
        strcpy(files[i], files[j]);
        strcpy(files[j],temp);
      }
    }
  } 

  // carrega cada corte na memória
  int ntoken = 0;
  for (int i = 0; i < num_files; i++){
    FILE *file = NULL;
    file = fopen(files[i], "r");
    if (!file){
      printf("loadCT: não conseguiu abrir arquivo\n");
      return(-2);
    }
    int nlines = 0;
    char *pbuf;
    char buf[MAX_LINE_SIZE] = {};
    while (pbuf = fgets(buf, sizeof(buf), file)){ // le a linha do arquivo
      char *p = pbuf;
      while ((p=strchr(pbuf, ',')) != NULL || (p=strchr(pbuf, '\n')) != NULL){  // obtem cada valor de pixel
        int len = p - pbuf;
        char token[MAX_TOKEN_SIZE];
        int k= 0;
        for (; k < len; k++){
          token[k] = pbuf[k];
        }
        token[k] = '\0';
        pbuf = p+1;
        imagem[ntoken++] = atoi(token);  
      }
      ++nlines;
    }
    fclose(file);      
  }  
  return(0);  
}

// *********************************************************************
// funcao para salvar os arquivos em disco
// *********************************************************************
int saveCT(int *imagem, int num_slices){
  
  int pixels_por_slice = WIDTH * WIDTH;
  
  char filename[MAX_FILENAME];
  char filepath[MAX_FILENAME];
  
  int cursor=0;
  
  for (int i = 0; i < num_slices; i++){
    snprintf(filename, 14, "/slice_%02d.txt", i);
    strcpy(filepath, IMAGE_OUTPUT_DIR);
    strcat(filepath, filename);
    // printf("%s\n", filepath); 
    FILE *fp;
   if ((fp = fopen(filepath,"w")) == NULL){
      return -1;
    }
    for (int j=cursor; j < (cursor + pixels_por_slice); j++){
      fprintf(fp, "%d", imagem[j]);
      if (((j+1) % WIDTH) == 0) {
        fprintf(fp, "\n");
      } else {
        fprintf(fp, ","); 
      }
    }
    cursor += pixels_por_slice;
    fclose(fp);
  }
   return 0;
}

// *********************************************************************
// obtém posição de um voxel no vetor linearizado
// *********************************************************************z
__host__ __device__ 
int getFlat(int x, int y, int z){
  int offset_y = WIDTH;
  int offset_z = WIDTH * WIDTH;  
  int flat = z * offset_z + y * offset_y + x;
  return flat;
}

// *********************************************************************
// obtém coordenadas de um elemento do vetor linearizado
// *********************************************************************
__host__ __device__ 
int getCoord(int flat, int *x, int *y, int *z){
  int offset_y = WIDTH;
  int offset_z = WIDTH * WIDTH;
  *z = flat / (offset_z);
  *y = (flat - ((*z) * (offset_z)))/offset_y;
  *x =  flat - ((*z) * (offset_z)) - ((*y) * offset_y);
  return 0;
}

// *********************************************************************
// calcula o pixel semente
// *********************************************************************
int calculateSeed(int *imagedata, int depth){
  // Inicialmente usando uma semente apenas.
  // Para identificar a semente incial utilizei o seguinte critério:
  // No corte central, busca na linha 255, a partir da coluna 255 o primeiro pixel entre -600 e -700 (tipicamente pulmão)
  int x = WIDTH / 2; //256
  int y = WIDTH / 2; // 256
  int z = depth / 2;
  int pos_seed = -1;
  for (int i = x; i < WIDTH; i++){
    int flat = getFlat(i, y, z);
    //printf("imagedata[%d] (%d, %d, %d): %d\n", flat, i, y, z, imagedata[flat]);
    if (imagedata[flat] > HU_PULMAO_MIN && imagedata[flat] < HU_PULMAO_MAX){
      pos_seed = flat;
      break;
    }
  }
  return (pos_seed);
}


// *********************************************************************
// funcao para verificar se é um pixel vizinho a região
// *********************************************************************
__host__ __device__ 
bool isNeighbor(int index, int *regiondata, int depth){
  int x; int y; int z;
  getCoord(index, &x, &y, &z);
  // printf("calcula feature: %d, %d, %d\n", x, y, z);
  for (int k = z-1; k <= z + 1; k++){
    for (int j = y-1; j <= y + 1; j++){
      for (int i = x-1; i <= x + 1; i++){
          if (((k > 0) && (k < depth)) && ((j > 0) && (j < WIDTH)) && ((i > 0) && (i < WIDTH))){ // testa se está dentro da imagem
            // printf("(k, j, i): (%d, %d, %d)\n", k, j, i);
            int index_neighbor = getFlat(i, j, k);
            if (index_neighbor != index)  // testa se não é o próprio elemento
              if (regiondata[index_neighbor] == 1)  // se um dos vizinhos é 1 ele é um vizinho
                return true;
          }
      }
    }
  }  
  return false;
}

// *********************************************************************
// MIN-MAX HU normalization
// *********************************************************************
__host__ __device__ 
float normalizeHU(int hu){
  if (hu<MIN_HU) 
    hu = MIN_HU;
  else if (hu > MAX_HU){
    hu = MAX_HU;
  } 
  return ((float)abs(hu-MIN_HU))/abs(MAX_HU-MIN_HU);
}

// *********************************************************************
// funcao para calcular o vetor de caracteristicas  (HU, MEAN, MIN, MAX, CVE)
// *********************************************************************
__host__ __device__ 
int calculateFeatures(int index, int *pixeldata, int depth, float *vector){
  vector[0] = normalizeHU(pixeldata[index]); //HU
  vector[1] = 0; // MEAN
  vector[2] = 0; // MIN
  vector[3] = 0; // MAX
  vector[4] = 0; // CVE (to be implemented)
  int x; int y; int z;
  getCoord(index, &x, &y, &z);
  // printf("calcula feature: %d, %d, %d\n", x, y, z);
  float min = 1;
  float max = 0;
  float sum = 0;
  float qtde = 0;
  for (int k = z-1; k <= z + 1; k++){
    for (int j = y-1; j <= y + 1; j++){
      for (int i = x-1; i <= x + 1; i++){
          if (((k > 0) && (k < depth)) && ((j > 0) && (j < WIDTH)) && ((i > 0) && (i < WIDTH))){
            // printf("(k, j, i): (%d, %d, %d)\n", k, j, i);
            float hu = normalizeHU(pixeldata[getFlat(i, j, k)]);
            sum+=hu;
            if (hu < min) min = hu;
            if (hu > max) max = hu;
            qtde++;
          }
      }
    }
  }
  vector[1] = (sum/qtde); // MEAN
  vector[2] = min; // MIN
  vector[3] = max; // MAX  
  return 0;
}

__device__
float calculateDistance(float *vector, float *seed){
  float sum = 0;
  for (int i = 0; i < NUM_FEATURES; i++){
    sum += pow((vector[i] - seed[i]), 2);
  }
  return (float)sqrt(sum);
}

__global__ 
void regionGrowing(int *imagedata, int *regiondata,  float *seed_vector, int *incluidos, int depth){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if ((x < WIDTH) && (y < WIDTH) && (z < depth)){
    int i = getFlat(x, y, z);
    if ((regiondata[i] != 1) && (isNeighbor(i, regiondata, depth))){
      float vector[NUM_FEATURES];
      calculateFeatures(i, imagedata, depth, vector);
      float distance = calculateDistance(vector, seed_vector);
      //printf("[hu, mean, min, max, cve]: [%f, %f, %f, %f, %f] :: distance:=%f\n", vector[0], vector[1], vector[2], vector[3], vector[4], distance );
      if (distance < LIMIAR){
        regiondata[i] = 1;
        *incluidos += 1;
      }
    }
  }
}

__global__ 
void regionMask(int *imagedata, int *regiondata, int depth){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if ((x < WIDTH) && (y < WIDTH) && (z < depth)){
    int i = getFlat(x, y, z);
    if  (regiondata[i] == 0) {
       regiondata[i] = MIN_HU;
    } else {
       regiondata[i] = imagedata[i];
    }   
  }
}

// *********************************************************************
// função principal do programa
// *********************************************************************
int main(void)
{
  
  // 1. inicializa variáveis no host
  int num_slices = 0;
  num_slices = countSlices();
  int num_elementos = num_slices * WIDTH * WIDTH;
  size_t sizect = num_elementos * sizeof(int);
  int *h_imagedata = (int *)malloc(sizect);
  int *h_regiondata = (int *)malloc(sizect);
  // inicializa vetor da regiao com zeros
  for (int i = 0; i < num_elementos; i++) h_regiondata[i] = 0;
  printf(">>> resumo da TC \n");  
  printf("num slices da TC: %d\n", num_slices);  
  printf("tamanho da TC (elementos): %d\n", num_elementos);
  printf("tamanho da TC (bytes): %lu\n", sizect);
  
  // 2. carrega os cortes na memoria principal
  printf(">>> carregando a tomografia na memória principal \n");  
  if (loadCT(h_imagedata) != 0){
    printf("erro ao carregar arquivos da tomografia\n");
    return(-1);
  }

  // 3. aloca as variaveis na memoria do device 
  int *d_imagedata;
  hipMalloc((void **)&d_imagedata, sizect);
  int *d_regiondata;
  hipMalloc((void **)&d_regiondata, sizect);  
  
  // 4. identifica o pixel semente e calcula vetor de caracteristicas (HU, MEAN, MIN, MAX, CVE)
  printf(">>> identificando a semente\n");    
  int index_seed = 0;
  if ((index_seed = calculateSeed(h_imagedata, num_slices)) < 0){
    printf("erro ao calcular o pixel semente\n");
    return(-1);
  }
  if (index_seed == 0){
     printf("não obteve a semente para o crescimento de região\n");
    return(-1);
  }else{
     printf("posição da semente: %d\n", index_seed);    
  }
  h_regiondata[index_seed] = 1;
  // calcula vetor de caracteristicas da semente(HU, MEAN, MIN, MAX, CVE)
  size_t size_vector = 5 * sizeof(float);
  float *h_seed_vector = (float *)malloc(size_vector);
  float *d_seed_vector;
  hipMalloc((void **)&d_seed_vector, size_vector);
  calculateFeatures(index_seed, h_imagedata, num_slices, h_seed_vector);
  
  // 5. copia os dados na memória do device
  hipMemcpy(d_imagedata, h_imagedata, sizect, hipMemcpyHostToDevice);
  hipMemcpy(d_regiondata, h_regiondata, sizect, hipMemcpyHostToDevice);
  hipMemcpy(d_seed_vector, h_seed_vector, size_vector, hipMemcpyHostToDevice);
             
  // 4. inicia loop com o crescimento de regiao e roda ate que novos pixels nao sejam mais incluidos
   int *h_incluidos = (int *)malloc(sizeof(int));
   int *d_incluidos;
   hipMalloc((void **)&d_incluidos, sizeof(int));
  
   // define o número de blocos e threads
   dim3 dimBlock(16, 16, 4);
   dim3 dimGrid(32, 32, (num_slices+4)/4); 
  int iteracao = 0;
  do{
      *h_incluidos = 0;
      hipMemcpy(d_incluidos, h_incluidos, sizeof(int), hipMemcpyHostToDevice);
      regionGrowing<<<dimGrid,dimBlock>>>(d_imagedata, d_regiondata, d_seed_vector, d_incluidos, num_slices);
      hipDeviceSynchronize();
      hipMemcpy(h_incluidos, d_incluidos, sizeof(int), hipMemcpyDeviceToHost);
      printf("%d) incluidos=%d\n", iteracao++, *h_incluidos); //debug
   } while(*h_incluidos != 0);

  // 5. Kernel que aplica uma máscara na imagem original para destacar a área obtida com o crescimento de região
  //    O resultado é armazenado na próxima mascara (d_regiondata)
  regionMask<<<dimGrid,dimBlock>>>(d_imagedata, d_regiondata, num_slices);
  
  // 6. copia resultado para memoria principal
  hipMemcpy(h_regiondata, d_regiondata, sizect, hipMemcpyDeviceToHost);
  
  // 7. salva em disco
  printf(">>> carregando a região em disco \n");  
  if (saveCT(h_regiondata, num_slices) != 0){
    printf("erro ao salvar o resultado em disco\n");
    return(-1);
  }
  
  // 8. limpeza
  free (h_imagedata);
  free (h_regiondata);
  free(h_seed_vector);
  free(h_incluidos);
  hipFree(d_imagedata);
  hipFree(d_regiondata);  
  hipFree(d_seed_vector);
  hipFree(d_incluidos);
  
  printf("Done\n");
  return 0;
}
